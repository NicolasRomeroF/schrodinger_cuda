
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void schroedinger(float * H2, float * H1, float * H0, float c, float dt, float dd){
	int blocksize = blockDim.y*blockDim.x;
	int blockId = gridDim.x*blockIdx.y + blockIdx.x;
	int tid = blockId*blocksize + blockDim.x*threadIdx.y + threadIdx.x;

	int tidDown = tid + blockDim.x;
	int tidUp = tid - blockDim.x;
	int tidRight = tid + blockDim.y;
	int tidLeft = tid - blockDim.y;
	H2[tid] = 2*H1[tid] - 2*H0[tid] + c*c*(dt/dd)*(dt/dd)*(H1[tidDown] + H1[tidUp] + H1[tidLeft] + H1[tidRight] - 4*H1[tid]);

}


__host__ int main(){
	dim3 blocksize;
	dim3 gridsize;

	float c = 1.0;
	float dt = 0.1;
	float dd = 2.0;


	int t = 300;
	int x = 256;
	int y = 256;
	float * H0 = (float*)malloc(sizeof(float*)*y*x);

	float * H1 = (float*)malloc(sizeof(float*)*y*x);
	
	float * H2 = (float*)malloc(sizeof(float*)*y*x);

	float * h0,* h1,* h2;
	hipMalloc(&h0, x*y*sizeof(float));
	hipMalloc(&h1, x*y*sizeof(float));
	hipMalloc(&h2, x*y*sizeof(float));

	hipMemcpy(h0,H0, x*y*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(h1,H1, x*y*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(h2,H2, x*y*sizeof(float), hipMemcpyHostToDevice);

	gridsize.x = x;
	gridsize.y = y;
	blocksize.x = 32;
	blocksize.y = 32;

	schroedinger<<<gridsize,blocksize>>>(h2,h1,h0,c,dt,dd);

	hipMemcpy(H2, h2, x*y*sizeof(float), hipMemcpyDeviceToHost);
	printf("%f ",h2[2]);

	return 0;
}